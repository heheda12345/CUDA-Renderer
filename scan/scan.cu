#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);


/* Helper function to round up to a power of 2. 
 */
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}


__global__ void local_sum(int* device_result, int* partial_result, int range) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int l = index * range, r = l + range;
    for (int i=l+1; i<r; i++)
        device_result[i] += device_result[i-1];
    partial_result[index] = device_result[r-1];
}

__global__ void forward(int* device_result, int der) {
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * der * 2;
    device_result[index + (der*2) - 1] += device_result[index + der - 1];
}

__global__ void backward(int* device_result, int der) {
    int l = (blockIdx.x * blockDim.x + threadIdx.x) * der * 2 + der - 1;
    int r = l + der;
    int t = device_result[l];
    device_result[l] = device_result[r];
    device_result[r] += t;
}

__global__ void set_to_zero(int *device_result, int index) {
    device_result[index] = 0;
}

__global__ void to_result(int* device_result, int* device_start, int* partial_result, int range, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int l = index * range, r = l + range;
    if (r != n)
        device_result[r]  = device_start[r-1] + partial_result[index];
    for (int i=r-1; i>l; i--)
        device_result[i] = device_start[i-1] + partial_result[index];
} 

void exclusive_scan(int* device_start, int length, int* device_result)
{
    /* Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the input and output in device memory,
     * but this is host code -- you will need to declare one or more CUDA 
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the input and the output arrays are sized to accommodate the next
     * power of 2 larger than the input.
     */
    int n = nextPow2(length);
    const int totalBlocks = min(n, 8192);
    int* partial_result;
    hipMalloc((void**)&partial_result, sizeof(int)*totalBlocks);
    int range = n/totalBlocks;
    local_sum<<<totalBlocks, 1>>>(device_start, partial_result, range);
    for (int der = 1; der < totalBlocks; der<<=1) {
        forward<<<totalBlocks/der/2, 1>>>(partial_result, der);
    }
    set_to_zero<<<1,1>>>(partial_result, totalBlocks-1);
    for (int der = totalBlocks/2; der >= 1; der >>= 1) {
        backward<<<totalBlocks/der/2, 1>>>(partial_result, der);
    }
    to_result<<<totalBlocks, 1>>>(device_result, device_start, partial_result, range, n);
    set_to_zero<<<1,1>>>(device_result, 0);
    hipFree(partial_result);
}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input; 
    // We round the array sizes up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness. 
    // You may have an easier time in your implementation if you assume the 
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), 
               hipMemcpyHostToDevice);

    // For convenience, both the input and output vectors on the device are
    // initialized to the input values. This means that you are free to simply
    // implement an in-place scan on the result vector if you wish.
    // If you do this, you will need to keep that fact in mind when calling
    // exclusive_scan from find_repeats.
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), 
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, end - inarray, device_result);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);
    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), 
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}

__global__ void find_repeat_pos(int* a, int *eq, int range, int length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int l = index * range, r = min(l + range, length);
    int cnt = 0;
    if (index == 0) {
        l++;
    }
    for (int i=l; i<r; i++)
        cnt += (a[i] == a[i-1]);
    eq[index] = cnt;
}

__global__ void copy_to_output(int* out, int* a, int* pos, int range, int length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int l = index * range, r = min(l + range, length);
    int id = pos[index];
    if (index == 0)
        l++;
    for (int i=l; i<r; i++)
        if (a[i] == a[i-1]) {
            out[id++] = i-1;
        }
    pos[index] = id;
}

int find_repeats(int *device_input, int length, int *device_output) {
    /* Finds all pairs of adjacent repeated elements in the list, storing the
     * indices of the first element of each pair (in order) into device_result.
     * Returns the number of pairs found.
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if 
     * it requires that. However, you must ensure that the results of
     * find_repeats are correct given the original length.
     */    
    int n = nextPow2(length);
    const int totalBlocks = min(n, 8192);
    int *eq_pos, *eq_idx;
    hipMalloc((void**)&eq_pos, sizeof(int)*totalBlocks);
    hipMalloc((void**)&eq_idx, sizeof(int)*totalBlocks);
    int range = n/totalBlocks;
    find_repeat_pos<<<totalBlocks, 1>>>(device_input, eq_pos, range, length);
    int* out = new int[totalBlocks];
    exclusive_scan(eq_pos, totalBlocks, eq_idx);
    copy_to_output<<<totalBlocks, 1>>>(device_output, device_input, eq_idx, range, length);
    int ret;
    hipMemcpy(&ret, &eq_idx[totalBlocks-1], sizeof(int), hipMemcpyDeviceToHost);
    hipFree(eq_pos);
    hipFree(eq_idx);
    return ret;
}

/* Timing wrapper around find_repeats. You should not modify this function.
 */
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), 
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    return endTime - startTime;
}

void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
